#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <malloc.h>
#include <stdio.h>
#include "Timing.h"
#include <math.h>

#define IMG_INPUT "input1.bmp"
//#define IMG_INPUT "input.bmp"
#define IMG_OUTPUT "output.bmp"

#define IMG_HEADER 1080

//#define IMG_WIDTH 4000
//#define IMG_HEIGHT 4000

#define IMG_WIDTH 1000
#define IMG_HEIGHT 1000


#define BLOCKDIM 1000

#define THREAD_X 32
#define THREAD_Y 32


__device__ const char SOBELX[] = { 1, 2, 1,  0, 0, 0,  -1, -2, -1};
__device__ const char SOBELY[] = { -1, 0, 1,  -2, 0, 2,  -1, 0, 1};
#pragma endregion


void DetectEdgesSequential(unsigned char *img, unsigned char *img2)
{
	const char SOBELX[] = { 1, 2, 1,  0, 0, 0,  -1, -2, -1 };
	const char SOBELY[] = { -1, 0, 1,  -2, 0, 2,  -1, 0, 1 };
	short newColor, x, y;
	int pos, pos2, pos3;
	for (int i = 1; i < IMG_HEIGHT -1; i++)
	{
		for (int j = 1; j < IMG_WIDTH -1; j++)
		{
			pos = IMG_HEIGHT * i + j;
			pos2 = pos - IMG_HEIGHT;
			pos3 = pos + IMG_HEIGHT;
			x =
				img[pos2 - 1] * SOBELX[0] +
				img[pos2] * SOBELX[1] +
				img[pos2 + 1] * SOBELX[2] +
				img[pos - 1] * SOBELX[3] +
				img[pos] * SOBELX[4] +
				img[pos + 1] * SOBELX[5] +
				img[pos3 - 1] * SOBELX[6] +
				img[pos3] * SOBELX[7] +
				img[pos3 + 1] * SOBELX[8];

			y =
				img[pos2 - 1] * SOBELY[0] +
				img[pos2] * SOBELY[1] +
				img[pos2 + 1] * SOBELY[2] +
				img[pos - 1] * SOBELY[3] +
				img[pos] * SOBELY[4] +
				img[pos + 1] * SOBELY[5] +
				img[pos3 - 1] * SOBELY[6] +
				img[pos3] * SOBELY[7] +
				img[pos3 + 1] * SOBELY[8];

			newColor = sqrt((float)(x * x + y * y));
			if (newColor > 255)
				newColor = 255;

			img2[pos] = newColor;
		}
	}
}

//nothing is shared
__global__ void DetectEdges(unsigned char *img, unsigned char *img2)
{
	int pos = blockIdx.x * blockDim.x + threadIdx.x;

	int pos2 = pos - IMG_WIDTH;
	int pos3 = pos + IMG_WIDTH;

	short x = 0;
	short y = 0;
	short newColor = 0;

	x =
		img[pos2 - 1] * SOBELX[0] +
		img[pos2] * SOBELX[1] +
		img[pos2 + 1] * SOBELX[2] +
		img[pos - 1] * SOBELX[3] +
		img[pos] * SOBELX[4] +
		img[pos + 1] * SOBELX[5] +
		img[pos3 - 1] * SOBELX[6] +
		img[pos3] * SOBELX[7] +
		img[pos3 + 1] * SOBELX[8];

	y =
		img[pos2 - 1] * SOBELY[0] +
		img[pos2] * SOBELY[1] +
		img[pos2 + 1] * SOBELY[2] +
		img[pos - 1] * SOBELY[3] +
		img[pos] * SOBELY[4] +
		img[pos + 1] * SOBELY[5] +
		img[pos3 - 1] * SOBELY[6] +
		img[pos3] * SOBELY[7] +
		img[pos3 + 1] * SOBELY[8];

	newColor = sqrt((float)(x * x + y * y));
	if (newColor > 255)
		newColor = 255;

	img2[pos] = newColor;
}


//only sobel operator is shared
__global__ void DetectEdgesSobelShared(unsigned char *img, unsigned char *img2)
{
	__shared__ char shr_sobelx[9];
	__shared__ char shr_sobely[9];

	int pos = (blockIdx.x) * blockDim.x + threadIdx.x;

	if (threadIdx.x < 9 && threadIdx.y == 0) {
		shr_sobelx[threadIdx.x] = SOBELX[threadIdx.x];
		shr_sobely[threadIdx.x] = SOBELY[threadIdx.x];
	}
	__syncthreads();

	int pos2 = pos - IMG_WIDTH;
	int pos3 = pos + IMG_WIDTH;

	short x = 0;
	short y = 0;
	x =
		img[pos2 - 1] * shr_sobelx[0] +
		img[pos2] * shr_sobelx[1] +
		img[pos2 + 1] * shr_sobelx[2] +
		img[pos - 1] * shr_sobelx[3] +
		img[pos] * shr_sobelx[4] +
		img[pos + 1] * shr_sobelx[5] +
		img[pos3 - 1] * shr_sobelx[6] +
		img[pos3] * shr_sobelx[7] +
		img[pos3 + 1] * shr_sobelx[8];
	y =
		img[pos2 - 1] * shr_sobely[0] +
		img[pos2] * shr_sobely[1] +
		img[pos2 + 1] * shr_sobely[2] +
		img[pos - 1] * shr_sobely[3] +
		img[pos] * shr_sobely[4] +
		img[pos + 1] * shr_sobely[5] +
		img[pos3 - 1] * shr_sobely[6] +
		img[pos3] * shr_sobely[7] +
		img[pos3 + 1] * shr_sobely[8];

	short newColor = sqrt((float)(x * x + y * y));

	if (newColor > 255)
		newColor = 255;

	img2[pos] = newColor;
}

__global__ void DetectEdgesSharedPixels2(unsigned char *img, unsigned char *img2)
{
	__shared__ unsigned char shr_img[THREAD_X + 2][THREAD_Y + 2];
	__shared__ char shr_sobelx[9];
	__shared__ char shr_sobely[9];

	int pos = blockIdx.x * THREAD_X * THREAD_Y + threadIdx.x + threadIdx.y * THREAD_Y; 

	char newTx = threadIdx.x + 1;
	char newTy = threadIdx.y + 1;

	shr_img[newTx][newTy] = img[pos];


	if (threadIdx.x < 9 && threadIdx.y == 0) {
		shr_sobelx[threadIdx.x] = SOBELX[threadIdx.x];
		shr_sobely[threadIdx.x] = SOBELY[threadIdx.x];
	}
	__syncthreads();

	if (newTx == 1)
		shr_img[0][newTy - 1] = img[pos + 1];
	else if (newTx == THREAD_X - 2)
		shr_img[THREAD_Y + 1][newTy + 1] = img[pos + 1];

	if (newTy == 1)
		shr_img[newTx - 1][0] = img[pos - IMG_WIDTH];
	else if (newTy == THREAD_Y - 2)
		shr_img[newTx + 1][THREAD_X + 1] = img[pos + IMG_WIDTH];

	if (newTx == 1 && newTy == 1)
		shr_img[newTx - 1][newTy - 1] = img[pos - IMG_WIDTH - 1];
	else if (newTx == THREAD_X - 1 && newTy == THREAD_Y - 1)
		shr_img[newTx + 1][newTy + 1] = img[pos + IMG_WIDTH + 1];
	else if (newTx == 1 && newTy == THREAD_Y - 1)
		shr_img[newTx - 1][newTy + 1] = img[pos + IMG_WIDTH - 1];
	else if (newTx == THREAD_X - 1 && newTy == 1)
		shr_img[newTx + 1][newTy - 1] = img[pos - IMG_WIDTH + 1];
	__syncthreads();

	short newColor = 0;
	short x = 0;
	short y = 0;

	x =
		shr_img[newTx - 1][newTy - 1] * shr_sobelx[0] +
		shr_img[newTx][newTy - 1] * shr_sobelx[1] +
		shr_img[newTx + 1][newTy - 1] * shr_sobelx[2] +
		shr_img[newTx - 1][newTy] * shr_sobelx[3] +
		shr_img[newTx][newTy] * shr_sobelx[4] +
		shr_img[newTx + 1][newTy] * shr_sobelx[5] +
		shr_img[newTx - 1][newTy + 1] * shr_sobelx[6] +
		shr_img[newTx][newTy + 1] * shr_sobelx[7] +
		shr_img[newTx + 1][newTy + 1] * shr_sobelx[8];

	y =
		shr_img[newTx - 1][newTy - 1] * shr_sobely[0] +
		shr_img[newTx][newTy - 1] * shr_sobely[1] +
		shr_img[newTx + 1][newTy - 1] * shr_sobely[2] +
		shr_img[newTx - 1][newTy] * shr_sobely[3] +
		shr_img[newTx][newTy] * shr_sobely[4] +
		shr_img[newTx + 1][newTy] * shr_sobely[5] +
		shr_img[newTx - 1][newTy + 1] * shr_sobely[6] +
		shr_img[newTx][newTy + 1] * shr_sobely[7] +
		shr_img[newTx + 1][newTy + 1] * shr_sobely[8];
	newColor = sqrt((float)(x * x + y * y));

	if (newColor > 255)
		newColor = 255;

	img2[pos] = newColor;
}


__global__ void DetectEdgesSharedPixels(unsigned char *img, unsigned char *img2)
{
	__shared__ unsigned char shr_img[BLOCKDIM * 3];
	__shared__ char shr_sobelx[9];
	__shared__ char shr_sobely[9];

	//int pos = blockIdx.x * blockDim.x + (threadIdx.x + threadIdx.y * blockDim.x);
	int pos = blockIdx.x * blockDim.x + threadIdx.x;

	int shr_pos = threadIdx.x + BLOCKDIM;
	int pos1 = shr_pos;

	shr_img[shr_pos - BLOCKDIM] = img[pos - IMG_WIDTH];
	shr_img[shr_pos] = img[pos];
	shr_img[shr_pos + BLOCKDIM] = img[pos + IMG_WIDTH];

	if (threadIdx.x < 9) {
		shr_sobelx[threadIdx.x] = SOBELX[threadIdx.x];
		shr_sobely[threadIdx.x] = SOBELY[threadIdx.x];
	}
	__syncthreads();


	short newColor = 0;
	int pos2 = shr_pos - BLOCKDIM;
	int pos3 = shr_pos + BLOCKDIM;

	short x = 0;
	short y = 0;

	if (threadIdx.x != 0 && threadIdx.x != blockDim.x - 1) {
		x =
			shr_img[pos2 - 1] * shr_sobelx[0] +
			shr_img[pos2] * shr_sobelx[1] +
			shr_img[pos2 + 1] * shr_sobelx[2] +
			shr_img[pos1 - 1] * shr_sobelx[3] +
			shr_img[pos1] * shr_sobelx[4] +
			shr_img[pos1 + 1] * shr_sobelx[5] +
			shr_img[pos3 - 1] * shr_sobelx[6] +
			shr_img[pos3] * shr_sobelx[7] +
			shr_img[pos3 + 1] * shr_sobelx[8];

		y =
			shr_img[pos2 - 1] * shr_sobely[0] +
			shr_img[pos2] * shr_sobely[1] +
			shr_img[pos2 + 1] * shr_sobely[2] +
			shr_img[pos1 - 1] * shr_sobely[3] +
			shr_img[pos1] * shr_sobely[4] +
			shr_img[pos1 + 1] * shr_sobely[5] +
			shr_img[pos3 - 1] * shr_sobely[6] +
			shr_img[pos3] * shr_sobely[7] +
			shr_img[pos3 + 1] * shr_sobely[8];
		newColor = sqrt((float)(x * x + y * y));

		if (newColor > 255)
			newColor = 255;
	}
	img2[pos] = newColor;
}


void DetectEdgesP(unsigned char *img, unsigned char *img2)
{
	unsigned char *d_img;
	unsigned char *d_img2;

	hipMalloc((void**)&d_img, sizeof(unsigned char)*IMG_WIDTH*IMG_HEIGHT);
	hipMalloc((void**)&d_img2, sizeof(unsigned char)*IMG_WIDTH*IMG_HEIGHT);

	hipMemcpy(d_img, img + IMG_HEADER, sizeof(unsigned char)*IMG_WIDTH*IMG_HEIGHT, hipMemcpyHostToDevice);

	//-------input.bmp--------
	//DetectEdges << <40000, 400 >> > (d_img, d_img2);			//179.42ms																							 //DoCalculating << < 40000, dim3(4000, 4000) >> > (d_img, d_num2); //Launch kernel
	//DetectEdgesSobelShared << <40000, 400>> > (d_img, d_img2);	 //194.28 ms
	//DetectEdgesSharedPixels << <16000, 1000 >> > (d_img, d_img2);		 //241.46 ms
	//DetectEdgesSharedPixels2 << <15625, dim3(THREAD_X,THREAD_Y) >> > (d_img, d_img2);		//998.52 ms

	//-------input1.bmp--------
	DetectEdges << <10000, 100 >> > (d_img, d_img2);			//15.8ms																							 //DoCalculating << < 40000, dim3(4000, 4000) >> > (d_img, d_num2); //Launch kernel
	//DetectEdgesSobelShared << <10000, 100>> > (d_img, d_img2);	 //16.73 ms
	//DetectEdgesSharedPixels << <10000, 100 >> > (d_img, d_img2);		 //19.45 ms
	//DetectEdgesSharedPixels2 << <976, dim3(THREAD_X,THREAD_Y) >> > (d_img, d_img2);		//67.67 ms

	hipMemcpy(img2 + IMG_HEADER, d_img2, sizeof(unsigned char)*IMG_WIDTH*IMG_HEIGHT, hipMemcpyDeviceToHost);
	hipFree(d_img);
}

void DetectEdgesS(unsigned char *img, unsigned char *img2) 
{
	unsigned char *d_img;
	unsigned char *d_img2;
	DetectEdgesSequential(img + IMG_HEADER, img2 + IMG_HEADER);
}


int main()
{
	unsigned char *img, *img2;
	FILE *f_input_img, *f_output_img;

	img = (unsigned char*)malloc(IMG_HEADER + sizeof(unsigned char) * IMG_WIDTH * IMG_HEIGHT);
	img2 = (unsigned char*)malloc(IMG_HEADER + sizeof(unsigned char) * IMG_WIDTH * IMG_HEIGHT);

	fopen_s(&f_input_img, IMG_INPUT, "rb");
	fread(img, 1, IMG_HEADER + IMG_WIDTH * IMG_HEIGHT, f_input_img);
	fclose(f_input_img);

	memcpy(img2, img, IMG_HEADER + IMG_HEIGHT * IMG_WIDTH);

	MEASURE_TIME(1, "EdgeDetection", DetectEdgesS(img, img2));

	fopen_s(&f_output_img, IMG_OUTPUT, "wb");
	fwrite(img2, 1, IMG_HEADER + IMG_WIDTH * IMG_HEIGHT, f_output_img);
	fclose(f_output_img);
	free(img);
}
